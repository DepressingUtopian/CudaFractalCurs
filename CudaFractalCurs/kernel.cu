#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <iomanip>
#include <fstream>
#include <string>
#include <sstream>
#include <vector>
#include <limits>
#include <thread>
#include <fstream>
// GLEW
#define GLEW_STATIC
#include <GL/glew.h>

// GLFW
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>

#include ""
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <thrust/complex.h>
#include <hip/hip_complex.h>

#include <sys/stat.h>

using namespace std;
// Function prototypes
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);

const GLuint WIDTH = 1920, HEIGHT = 1080;

GLdouble screen_ratio = (double)WIDTH / (double)HEIGHT;

double cx = 0.0, cy = 0.0, zoom = 1.0;
int fps = 0;
bool isChange = false;
GLFWwindow *window = nullptr;
GLuint shaderProgram;

const GLchar* vertexShaderSource = "#version 330 core\n"
"layout (location = 0) in vec3 position;\n"
"layout (location = 1) in vec3 color;\n"
"out vec3 ourColor;\n"
"void main()\n"
"{\n"
"gl_Position = vec4(position, 1.0);\n"
"ourColor = color;\n"
"}\0";
const GLchar* fragmentShaderSource = "#version 330 core\n"
"in vec3 ourColor;\n"
"out vec4 color;\n"
"void main()\n"
"{\n"
"color = vec4(ourColor, 1.0f);\n"
"}\n\0";

GLuint MAX_ITERATION = 1;

float elapsed_time_gpu = 0;

dim3 blocks, threads;

//GLuint vbo; //int ��������� �� Vertex Buffer Object

GLuint VBO;
GLuint VAO;

struct hipGraphicsResource *cuda_vbo_resource; //��������� ��������������� ���������� VBO � CUDA


double last_time = 0, current_time = 0;
unsigned int ticks = 0;

bool keys[1024] = { 0 };


static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



static void cursor_callback(GLFWwindow* window, double xpos, double ypos)
{
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods)
{
	double xpos, ypos;
	glfwGetCursorPos(window, &xpos, &ypos);

	double xr = 2.0 * (xpos / (double)WIDTH - 0.5);
	double yr = 2.0 * (ypos / (double)HEIGHT - 0.5);

	if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS) {
		cx += (xr - cx) / zoom / 2.0;
		cy -= (yr - cy) / zoom / 2.0;
		isChange = true;
	}
	//isChange = true;
}

void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
	zoom += yoffset * 0.1 * zoom;
	if (zoom < 0.1) {
		zoom = 0.1;
		
	}
	isChange = true;
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	const double d = 0.1 / zoom;

	if (action == GLFW_PRESS) {
		keys[key] = true;
		isChange = true;
	}
	else if (action == GLFW_RELEASE) {
		keys[key] = false;
		isChange = true;
	}

	if (keys[GLFW_KEY_ESCAPE]) {
		glfwSetWindowShouldClose(window, 1);

	}
	else if (keys[GLFW_KEY_A]) {
		cx -= d;
		isChange = true;
	}
	else if (keys[GLFW_KEY_D]) {
		cx += d;
		isChange = true;
	}
	else if (keys[GLFW_KEY_W]) {
		cy += d;
		isChange = true;
	}
	else if (keys[GLFW_KEY_S]) {
		cy -= d;
		isChange = true;
	}
	else if (keys[GLFW_KEY_MINUS] &&
		MAX_ITERATION < std::numeric_limits <int>::max() - 10) {
		MAX_ITERATION += 10;
	}
	else if (keys[GLFW_KEY_EQUAL]) {
		MAX_ITERATION -= 10;
		if (MAX_ITERATION <= 0) {
			MAX_ITERATION = 0;
		}
	}
	
}
static void update_window_title()
{
	std::ostringstream ss;
	ss << "Mandelbrot Renderer";
	ss << ", FPS: " << fps;
	ss << ", Iterations: " << MAX_ITERATION;
	ss << ", Zoom: " << zoom;
	ss << ", At: (" << std::setprecision(8) << cx << " + " << cy << "i)";
	glfwSetWindowTitle(window, ss.str().c_str());
	//isChange = true;
}


__device__ double mapToReal(int x, int windowWidth, double minR, double maxR)
{
	double range = maxR - minR;
	return x * (range / windowWidth) + minR;
}
__device__ double mapToImaginary(int y, int windowHeigth, double minI, double maxI)
{
	double range = maxI - minI;
	return y * (range / windowHeigth) + minI;
}
__device__ double MandelbrotFunction(double p, double q, int maxIteration)
{
	int i = 0;
	double x_t = 0.0, y_t = 0.0;
	while (i++ < maxIteration && x_t*x_t + y_t * y_t < 4.0)
	{
		double temp = x_t * x_t - y_t * y_t + p;
		y_t = 2.0 * x_t * y_t + q;
		x_t = temp;

	}
	return (double)i / (double)maxIteration;
}

void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags)
{
	unsigned int size = WIDTH * HEIGHT * sizeof(float) * 6;

	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);
	glBufferData(GL_ARRAY_BUFFER, size, NULL, GL_DYNAMIC_DRAW);

	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	//	glVertexAttribPointer(1, 4, sizeof(uchar4), GL_FALSE,size, (GLvoid*)0);
	//	glEnableVertexAttribArray(0);
}

void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{
	HANDLE_ERROR(hipGraphicsUnregisterResource(cuda_vbo_resource));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}
__device__ void setCoord(int x, int y, int offset,float *pixels)
{
	pixels[offset * 6] = (-1.0f + 2.0f * (float)(x / (float)WIDTH));			//x
	pixels[offset * 6 + 1] = (-1.0f + 2.0f * (float)(y / (float)HEIGHT));		//y
	pixels[offset * 6 + 2] = 0.0f;	//z

}
__device__ void setColor(int offset, double t,float *pixels,int maxIteration)
{
	//float t = (float)(ColorValue % maxIteration);
	

	pixels[offset * 6 + 3] = 9.0 * (1.0 - t) * t * t * t;
	pixels[offset * 6 + 4] = 15.0 * (1.0 - t) * (1.0 - t) * t * t;
	pixels[offset * 6 + 5] = 8.5 * (1.0 - t) * (1.0 - t) * (1.0 - t) * t;
	
}
__global__ void MandelbrotKernel(float* screen, int windowHeigth, int windowWidth, int maxIteration,double zoom, double cx, double cy, double screen_ratio)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	

	if (x >= WIDTH || y >= HEIGHT)
		return;

	int offset = x + y * WIDTH;
	//double p = mapToReal(x, windowWidth, minR, maxR);
	//double q = mapToImaginary(y, windowHeigth, minI, maxI);
	double p = (double)screen_ratio * (double)((double)x / (double)WIDTH - 0.5);
	double q = ((double)y / (double)HEIGHT - 0.5);
	p /= zoom;
	q /= zoom;
	p += cx;
	q += cy;
	double MandelbrotValue = MandelbrotFunction(p, q, maxIteration);
	__syncthreads();
	setCoord(x, y, offset, screen);	
	setColor(offset, MandelbrotValue, screen,maxIteration);

}

void InitOpenGL()
{
	//������������� GLFW
	glfwInit();


	//��������� GLFW
	//�������� ����������� ��������� ������ OpenGL. 
	//�������� 
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	//��������
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	//��������� �������� ��� �������� ��������� ��������
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	//���������� ����������� ��������� ������� ����
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);
}
void initCuda(int deviceId) {
	int deviceCount = 0;
	HANDLE_ERROR(hipGetDeviceCount(&deviceCount));

	if (deviceCount <= 0) {
		printf("No CUDA devices found\n");
		exit(-1);
	}

	HANDLE_ERROR(cudaGLSetGLDevice(deviceId));

	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceProperties(&properties, deviceId));

	threads.x = 32;
	threads.y = properties.maxThreadsPerBlock / threads.x - 2; // to avoid hipErrorLaunchOutOfResources error

	blocks.x = (WIDTH + threads.x - 1) / threads.x;
	blocks.y = (HEIGHT + threads.y - 1) / threads.y;

	printf(
		"Debug: blocks(%d, %d), threads(%d, %d)\nCalculated Resolution: %d x %d\n",
		blocks.x, blocks.y, threads.x, threads.y, blocks.x * threads.x,
		blocks.y * threads.y);
}

void MaldelbrotGPU_Calculation()
{
	float *dev_screen;
	//float * gpu_PixelsMemory = (float*)malloc(WIDTH * HEIGHT * sizeof(float) * 6);
	dim3 *blocks2 = new dim3(WIDTH,HEIGHT);
	
	size_t size;

	HANDLE_ERROR(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
	HANDLE_ERROR(
		hipGraphicsResourceGetMappedPointer((void**)&dev_screen, &size, cuda_vbo_resource));
	hipEvent_t startEvent, stopEvent;
	float elapsedTime = 0.0f;
	HANDLE_ERROR(hipEventCreate(&startEvent));
	HANDLE_ERROR(hipEventCreate(&stopEvent));
	HANDLE_ERROR(hipEventRecord(startEvent, 0));

	// Render Image
	MandelbrotKernel << <blocks, threads >> > (dev_screen, HEIGHT, WIDTH, MAX_ITERATION,zoom,cx,cy,screen_ratio);
	HANDLE_ERROR(hipDeviceSynchronize());
	/*
	hipMemcpy(gpu_PixelsMemory, dev_screen, WIDTH * HEIGHT * sizeof(float) * 6, hipMemcpyDeviceToHost);
	int count = 0;
	for (int i = 0; i < WIDTH * HEIGHT * 6; i++)
	{
		if (i >= WIDTH * HEIGHT * 6 - WIDTH)
		{


			if (count == 6)
			{
				std::cout << std::endl;
				count = 0;
			}
			std::cout << " " << i << " " << gpu_PixelsMemory[i] << " ";
			count++;
		}
	}*/
	
	// Kernel Time measure
	HANDLE_ERROR(hipEventRecord(stopEvent, 0));
	HANDLE_ERROR(hipEventSynchronize(stopEvent));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
	HANDLE_ERROR(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));

//	std::cout << std::endl;
	printf("����� ����������: %f s\n", elapsedTime / 1000.0);
	elapsed_time_gpu = elapsedTime / 1000.0;
	
}

void InitShaders(GLuint &prog)
{
	GLint success;
	GLchar infoLog[512];

	GLuint vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
	glCompileShader(vertexShader);
	// Check for compile time errors

	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(vertexShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::VERTEX::COMPILATION_FAILED\n" << infoLog << std::endl;
	}
	// Fragment shader
	GLuint fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
	glCompileShader(fragmentShader);
	// Check for compile time errors
	glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &success);
	if (!success)
	{
		glGetShaderInfoLog(fragmentShader, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n" << infoLog << std::endl;
	}
	// Link shaders
	prog = glCreateProgram();
	glAttachShader(prog, vertexShader);
	glAttachShader(prog, fragmentShader);
	glLinkProgram(prog);
	// Check for linking errors
	glGetProgramiv(prog, GL_LINK_STATUS, &success);
	if (!success) {
		glGetProgramInfoLog(prog, 512, NULL, infoLog);
		std::cout << "ERROR::SHADER::PROGRAM::LINKING_FAILED\n" << infoLog << std::endl;
	}
	glDeleteShader(vertexShader);
	glDeleteShader(fragmentShader);
}

// The MAIN function, from here we start the application and run the game loop

int main()
{

	setlocale(LC_ALL, "Russian");
	ofstream time;
	ofstream iters;
	time.open("./time.txt", ios::out | ios::trunc);
	iters.open("./iters.txt", ios::out | ios::trunc);
	//std::thread thread(MaldelbrotGPU_Calculation);

	// Init GLFW
	glfwInit();
	// Set all the required options for GLFW
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 1);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	atexit(glfwTerminate);


	// Create a GLFWwindow object that we can use for GLFW's functions
	window = glfwCreateWindow(WIDTH, HEIGHT, "MandelbrotCuda", nullptr, nullptr);

	// Set the required callback functions
	
	glfwSetKeyCallback(window, key_callback);
	glfwSetCursorPosCallback(window, cursor_callback);
	glfwSetMouseButtonCallback(window, mouse_button_callback);
	glfwSetScrollCallback(window, scroll_callback);
	glfwSetInputMode(window, GLFW_CURSOR_NORMAL, GLFW_STICKY_KEYS);
	
	glfwMakeContextCurrent(window);

	// Set this to true so GLEW knows to use a modern approach to retrieving function pointers and extensions
	glewExperimental = GL_TRUE;
	// Initialize GLEW to setup the OpenGL Function pointers
	glewInit();

	// Define the viewport dimensions
	glViewport(0, 0, WIDTH, HEIGHT);


	// Build and compile our shader program
	// Vertex shader
	


	// Set up vertex data (and buffer(s)) and attribute pointers
	GLfloat vertices[] = {
		// Positions         // Colors
		 0.5f, -0.5f, 0.0f,  1.0f, 0.0f, 0.0f,  // Bottom Right
		-0.5f, -0.5f, 0.0f,  0.0f, 1.0f, 0.0f,  // Bottom Left
		 0.0f,  0.5f, 0.0f,  0.0f, 0.0f, 1.0f   // Top 
	};
	initCuda(0);
	
	GLuint VBO, VAO;
	createVBO(&VBO, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);
	MaldelbrotGPU_Calculation();


	InitShaders(shaderProgram);

	glGenVertexArrays(1, &VAO);
	

	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, VBO);


	// Position attribute
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(GLfloat), (GLvoid*)0);
	glEnableVertexAttribArray(0);
	// Color attribute
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 6 * sizeof(GLfloat), (GLvoid*)(3 * sizeof(GLfloat)));
	glEnableVertexAttribArray(1);

	glBindVertexArray(0); // Unbind VAO
	last_time = glfwGetTime();
	// Game loop
	while (!glfwWindowShouldClose(window))
	{
		
	    MaldelbrotGPU_Calculation();
			
		
		// Check if any events have been activiated (key pressed, mouse moved etc.) and call corresponding response functions
		glfwPollEvents();
		//MaldelbrotCPU_Calculation();
		// Render
		// Clear the colorbuffer
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
		

		// Draw the triangle
		glUseProgram(shaderProgram);
		glBindVertexArray(VAO);
		glDrawArrays(GL_POINTS, 0, HEIGHT * WIDTH);
		glBindVertexArray(0);

		// Swap the screen buffers
		glfwSwapBuffers(window);

		ticks++;
		current_time = glfwGetTime();
		if (current_time - last_time > 1.0) {
			fps = ticks;
			update_window_title();
			last_time = glfwGetTime();
			ticks = 0;
		}
		if (MAX_ITERATION < 200)
		{

			iters << MAX_ITERATION << endl;
			time << elapsed_time_gpu << endl;
			MAX_ITERATION++;
		}
	}
	// Properly de-allocate all resources once they've outlived their purpose
	glDeleteVertexArrays(1, &VAO);
	glDeleteBuffers(1, &VBO);
	// Terminate GLFW, clearing any resources allocated by GLFW.
	glfwTerminate();
	return 0;
}

